
#include <hip/hip_runtime.h>
#include <assert.h>
#include <iostream>

/*
  Tiled 2d Convolution implementation using shared and constant memory
  @param A : input matrix
  @param widht : width of A
  @param height : height of A
  @param F : Filter
  @param MASK_SIZE : dimension of F
*/

const int TILE_SIZE = 16;
const int MASK_SIZE = 5;

__global__ void convolution_2d_tiled(float *A, int width, 
                                     int height, float *F, float *P) {

  int const MASK_OFFSET = MASK_SIZE/2;

  __shared__ float i_shmem[TILE_SIZE + MASK_SIZE-1][TILE_SIZE + MASK_SIZE-1];

  // local position of thread inside the block
  int tx = threadIdx.x;
  int ty = threadIdx.y;

  // boundaries
  int o_row = blockIdx.x * TILE_SIZE + tx;
  int o_col = blockIdx.y * TILE_SIZE + ty;

  // position of the thread in the input space
  int i_row = o_row - MASK_OFFSET;
  int i_col = o_col - MASK_OFFSET;

  if ((i_row >= 0 && i_row < height)
  && (i_col >= 0 && i_col < width))
    i_shmem[tx][ty] = A[i_row * width + i_col];
  else
    i_shmem[tx][ty] = 0.0f;

  // wait that all threads have loaded into shmem
  __syncthreads();

  float tmp = 0.0f;

    if (tx < TILE_SIZE && ty < TILE_SIZE) {
      // compute output element
      for (int i = 0; i < MASK_SIZE; i++) {
        for (int j = 0; j < MASK_SIZE; j++) {
          tmp += F[i*MASK_SIZE+j] * i_shmem[i+ty][j+tx];
        }
      }

      if(o_row < height && o_col < width){
        P[o_row * width + o_col] = tmp;
      }
    }
}

int main () {

  const int N = 16;

  float A[N][N], C[N][N], F[MASK_SIZE][MASK_SIZE];
  float *c_A, *c_F, *c_C;

  int a_size = sizeof(float) * N * N;
  int f_size = sizeof(float) * MASK_SIZE * MASK_SIZE;

  // allocate memory on device
  hipMalloc((void**)&c_A,a_size);
  hipMalloc((void**)&c_F,f_size);
  hipMalloc((void**)&c_C,a_size);

  for(int j=0; j < N; j++) {
    for(int k=0; k < N; k++) { A[j][k] = 1; C[j][k] = 0; }}

  for(int j=0; j < MASK_SIZE; j++) {
          for(int k=0; k < MASK_SIZE; k++) { F[j][k] = 1; }}

  // copy from host
  hipMemcpy(c_A,&A,a_size,hipMemcpyHostToDevice);
  hipMemcpy(c_F,&F,f_size,hipMemcpyHostToDevice);
  hipMemcpy(c_C,&C,a_size,hipMemcpyHostToDevice);

  dim3 block(TILE_SIZE + MASK_SIZE - 1,TILE_SIZE + MASK_SIZE - 1);
  dim3 grid(N/TILE_SIZE + 1, N/TILE_SIZE + 1);

  convolution_2d_tiled<<<grid,block>>>(c_A, N, N, c_F, c_C);

  // transfer back on host from device
  hipMemcpy(C,c_C,a_size,hipMemcpyDeviceToHost);

  for(int j=0; j < N; j++) {
    for(int k=0; k < N; k++) {
      std::cout << C[j][k] << " ";
    }
    std::cout << std::endl;
  }


}