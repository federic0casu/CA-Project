
#include <hip/hip_runtime.h>
#include <assert.h>
#include <iostream>

// baseline 2d convolution
// Only use odd kernel sizes
__global__ void convolution_2d(int *A,int *F, int p, int n, int *C) {

  int tmp = 0;

  int tx = threadIdx.x;
  int ty = threadIdx.y;

  int row = blockIdx.x*blockDim.x + tx;
  int col = blockIdx.y*blockDim.y + ty;

  int offset_k = p/2;

  int row_offset = row - offset_k;
  int col_offset = col - offset_k;

  for(int kx = 0; kx < p; kx++) {
    for(int ky = 0; ky < p; ky++) {
      if(row_offset + kx >= 0 && row_offset + kx < n) {
        if(col_offset + ky >= 0 && col_offset + ky < n)
          tmp += A[(row_offset + kx)*n + col_offset + ky]
                * F[kx * p + ky];
      }
    }
  }

  if(row < n && col < n){
    C[row*n + col] = tmp;
  }
}

int main () {

  const int WARP_SIZE = 32;

  int N = 1024;

  const int MASK_SIZE = 3;
  const int BLOCK_SIZE = 32;

  int *A,*C,*F;
  int *c_A, *c_F, *c_C;

  int a_size = sizeof(int) * N * N;
  int f_size = sizeof(int) * MASK_SIZE * MASK_SIZE;

  A = (int*)malloc(a_size);
  F = (int*)malloc(f_size);
  C = (int*)malloc(a_size);

  // allocate memory on device
  hipMalloc((void**)&c_A,a_size);
  hipMalloc((void**)&c_F,f_size);
  hipMalloc((void**)&c_C,a_size);

  for(int j = 0; j < N; j++) {
    for(int k = 0; k < N; k++) {
      A[j*N+k] = 1;
      C[j*N+k] = 0;
    }
  }

  for(int j=0; j < MASK_SIZE; j++) {
    for(int k=0; k < MASK_SIZE; k++) {
      F[j*MASK_SIZE+k] = 1;
    }
  }

  hipMemcpy(c_A,&A,a_size,hipMemcpyHostToDevice);
  hipMemcpy(c_F,&F,f_size,hipMemcpyHostToDevice);
  hipMemcpy(c_C,&C,a_size,hipMemcpyHostToDevice);

  dim3 block(BLOCK_SIZE,BLOCK_SIZE);
  dim3 grid(N/BLOCK_SIZE+1,N/BLOCK_SIZE+1);

  convolution_2d<<<grid,block>>>(c_A, c_F, MASK_SIZE, N, c_C);

  // transfer back on host from device
  hipMemcpy(C,c_C,a_size,hipMemcpyDeviceToHost);

  free(A);
  free(F);
  free(C);

  hipFree(c_A);
  hipFree(c_F);
  hipFree(c_C);

}